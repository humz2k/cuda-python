
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

extern "C"{
    void test(int a){
        printf("Hello World! I got value %d.\n",a);
    }
}


extern "C"{
    void other(float* test, int* dims, int ndims){
        printf("I am reading a numpy array with %d dimensions: (",ndims);
        for (int i = 0; i < ndims; i++){
            printf("%d",dims[i]);
            if (i != (ndims-1)){
                printf(", ");
            }
        }
        printf(")\n");
    }
}