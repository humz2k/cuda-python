
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void within_radius_kernel(const float3* __restrict d_particles, int* __restrict out, float3 center, float radius, int n_particles){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n_particles){
        return;
    }
    float3 my_particle = d_particles[i];
    my_particle.x -= center.x;
    my_particle.y -= center.y;
    my_particle.z -= center.z;

    float r2 = my_particle.x*my_particle.x + my_particle.y*my_particle.y + my_particle.z*my_particle.z;
    float r = sqrtf(r2);
    out[i] = r < radius;

}

extern "C"{
    void within_radius_binding(float3* h_particles, int* h_out, int n_particles, float x, float y, float z, float radius){
        float3* d_particles; hipMalloc(&d_particles,sizeof(float3)*n_particles);
        hipMemcpy(d_particles,h_particles,sizeof(float3)*n_particles,hipMemcpyHostToDevice);
        int* d_out; hipMalloc(&d_out,sizeof(int)*n_particles);
        
        int blockSize = 64;
        int numBlocks = (n_particles + (blockSize - 1)) / blockSize;

        within_radius_kernel<<<numBlocks,blockSize>>>(d_particles,d_out,make_float3(x,y,z),radius,n_particles);

        hipDeviceSynchronize();

        hipMemcpy(h_out,d_out,sizeof(int)*n_particles,hipMemcpyDeviceToHost);

        hipFree(d_particles);
        hipFree(d_out);
    }
}