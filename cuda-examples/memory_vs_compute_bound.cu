#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "cputimer.hpp"

__global__ void memory_bound(const float4* __restrict my_data, float4* __restrict my_output, int n){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float4 me = my_data[i];
    my_output[i] = me;
}

__global__ void compute_bound(const float4* __restrict my_data, float4* __restrict my_output, int n){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float4 me = my_data[i];
    for (int i = 0; i < 10000; i++){
        me.x = sqrtf(me.x);
        me.y = sqrtf(me.y);
        me.z = sqrtf(me.z);
        me.w = sqrtf(me.w);
    }
    my_output[i] = me;
}

int main(){
    return 0;
}