
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mySimpleKernel(int* d_my_array, int n){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n){
        return;
    }
    d_my_array[i] = d_my_array[i] + 1;
}

extern "C"{
    void mySimpleKernelPythonBindings(int* h_my_array, int n){
        int* d_my_array; hipMalloc(&d_my_array,sizeof(int)*n);
        hipMemcpy(d_my_array,h_my_array,sizeof(int)*n,hipMemcpyHostToDevice);
        
        int blockSize = 64;
        int numBlocks = (n + (blockSize - 1)) / blockSize;

        mySimpleKernel<<<numBlocks,blockSize>>>(d_my_array,n);

        hipDeviceSynchronize();

        hipMemcpy(h_my_array,d_my_array,sizeof(int)*n,hipMemcpyDeviceToHost);

        hipFree(d_my_array);
    }
}